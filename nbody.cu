 

#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
// #include <stdlib.h>
// #include <omp.h>
//#include "timer.h" // Include the timer header
//#include <cuda_runtime.h>
// #include "matric.h" // Include your custom matric.h header
// #include <stdio.h>
// #include <assert.h>


#define SOFTENING 1e-9f

typedef struct {
    float x, y, z, vx, vy, vz;
} Body;

// Macro definitions
//#define THROUGHPUT(operations, seconds) ((operations) / (seconds) / 1e9) // GOPS
//#define RATIO_TO_PEAK_BANDWIDTH(actual_bandwidth, peak_bandwidth) ((actual_bandwidth) / (peak_bandwidth))

void randomizeBodies(float *data, int n) {
    for (int i = 0; i < n; i++) {
        data[i] = 2.0f * (rand() / (float)RAND_MAX) - 1.0f;
    }
}

__global__ void bodyForce(Body *p, float dt, int n, float *Fx, float *Fy, float *Fz) {


    // for (int i = 0; i < n; i++) {
         int i = threadIdx.x + blockIdx.x * blockDim.x;
         if (i<n){
            Fx[i] = 0.0f;
            Fy[i] = 0.0f;
            Fz[i] = 0.0f;

            for (int j = 0; j < n; j++) {
                if (i != j) {
                    float dx = p[j].x - p[i].x;
                    float dy = p[j].y - p[i].y;
                    float dz = p[j].z - p[i].z;
                    float distSqr = dx * dx + dy * dy + dz * dz + SOFTENING;
                    float invDist = 1.0f / sqrtf(distSqr);
                    float invDist3 = invDist * invDist * invDist;

                    Fx[i] += dx * invDist3;
                    Fy[i] += dy * invDist3;
                    Fz[i] += dz * invDist3;
                    //printf("stampa f: %d\n",Fx[i]);
                }
            }

            p[i].vx += dt * Fx[i];
            p[i].vy += dt * Fy[i];
            p[i].vz += dt * Fz[i];
    // }
         }
}

void saveForcesToFile(const char *filename, int nBodies, Body *p, float *Fx, float *Fy, float *Fz) {
    FILE *file = fopen(filename, "w");
    if (!file) {
        fprintf(stderr, "Unable to open file %s for writing.\n", filename);
        return;
    }
    for (int i = 0; i < nBodies; i++) {
        fprintf(file, "Body %d: x = %.3f, y = %.3f, z = %.3f, Fx = %.3f, Fy = %.3f, Fz = %.3f\n",
                i, p[i].x, p[i].y, p[i].z, Fx[i], Fy[i], Fz[i]);
    }
    fclose(file);
}

__global__ void integration(int n, Body* p_d, float dt){
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i<n){
        p_d[i].x += p_d[i].vx * dt;
        p_d[i].y += p_d[i].vy * dt;
        p_d[i].z += p_d[i].vz * dt;
    }
}

int main(int argc, char **argv) {

    int nBodies = 30000;
    if (argc > 1) nBodies = atoi(argv[1]);

    const float dt = 0.01f; // time step
    const int nIters = 500;  // simulation iterations

    int bytes = nBodies * sizeof(Body);
    Body *p_h ; //= (Body *)malloc(bytes);
    Body *p_d ; //= (Body *)malloc(bytes);



    hipHostMalloc(&p_h,bytes, hipHostMallocDefault);
    hipMalloc(&p_d, bytes);

    if (p_h == NULL || p_h == NULL) {
        fprintf(stderr, "Unable to allocate memory for bodies.\n");
        return 1;
    }

    float *buf = (float *)malloc(6 * nBodies * sizeof(float));
    if (buf == NULL) {
        fprintf(stderr, "Unable to allocate memory for buffer.\n");
        hipFree(p_h);
        return 1;
    }

    size_t threads_per_blocks = 256;
    size_t number_of_blocks = (nBodies + threads_per_blocks - 1) / threads_per_blocks;

    randomizeBodies(buf, 6 * nBodies); // Init pos / vel data
    for (int i = 0; i < nBodies; i++) {
        p_h[i].x = buf[6 * i];
        p_h[i].y = buf[6 * i + 1];
        p_h[i].z = buf[6 * i + 2];
        p_h[i].vx = buf[6 * i + 3];
        p_h[i].vy = buf[6 * i + 4];
        p_h[i].vz = buf[6 * i + 5];
    }

    free(buf);

    // float *Fx = (float *)malloc(nBodies * sizeof(float));
    // float *Fy = (float *)malloc(nBodies * sizeof(float));
    // float *Fz = (float *)malloc(nBodies * sizeof(float));
    float *Fx_h ;     float *Fx_d ;
    float *Fy_h ;     float *Fy_d ;
    float *Fz_h ;     float *Fz_d ;

    hipHostMalloc(&Fx_h,nBodies * sizeof(float), hipHostMallocDefault);   hipHostMalloc(&Fy_h,nBodies * sizeof(float), hipHostMallocDefault);       hipHostMalloc(&Fz_h,nBodies * sizeof(float), hipHostMallocDefault);
    hipMalloc(&Fx_d, nBodies * sizeof(float));      hipMalloc(&Fy_d, nBodies * sizeof(float));          hipMalloc(&Fz_d, nBodies * sizeof(float));

    if (Fx_h == NULL || Fy_h == NULL || Fz_h == NULL) {
        fprintf(stderr, "Unable to allocate memory for force arrays.\n");
        hipHostFree(p_h);
        if (Fx_h) hipHostFree(Fx_h);
        if (Fy_h) hipHostFree(Fy_h);
        if (Fz_h) hipHostFree(Fz_h);
        return 1;
    }

    float totalTime = 0.0;
    hipMemcpy(p_d,p_h,bytes,hipMemcpyHostToDevice);

    hipEvent_t start,stop;
    float time;


    for (int iter = 1; iter <= nIters; iter++) {
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start,0);

        bodyForce<<<number_of_blocks, threads_per_blocks>>>(p_d, dt, nBodies, Fx_d, Fy_d, Fz_d); // compute interbody forces
        hipDeviceSynchronize();

        integration<<<number_of_blocks, threads_per_blocks>>>(nBodies,p_d,dt);
        hipDeviceSynchronize();

        hipEventRecord(stop,0);
        hipEventSynchronize(stop);
        if (iter > 1) { // First iter is warm up
            hipEventElapsedTime(&time,start,stop);
            totalTime += time/1000;
        }
        //printf("Iteration %d: %.3f seconds\n", iter, time);
        if (iter==nIters-1) printf("Final Iteration %d: %.3f seconds\n", totalTime);
    }

    hipMemcpy(p_h,p_d,bytes,hipMemcpyDeviceToHost);
    hipMemcpy(Fx_h,Fx_d,nBodies * sizeof(float),hipMemcpyDeviceToHost);
    hipMemcpy(Fy_h,Fy_d,nBodies * sizeof(float),hipMemcpyDeviceToHost);
    hipMemcpy(Fz_h,Fz_d,nBodies * sizeof(float),hipMemcpyDeviceToHost);
    saveForcesToFile("forces.txt", nBodies, p_h, Fx_h, Fy_h, Fz_h);

    double avgTime = totalTime / (double)(nIters - 1);
    double rate = (double)nBodies / avgTime;

    printf("Average rate for iterations 2 through %d: %.3f steps per second.\n",
           nIters, rate);
    printf("%d Bodies: average %0.3f Billion Interactions / second\n", nBodies, 1e-9 * nBodies * nBodies / avgTime);

    hipHostFree(p_h);
    hipFree(p_d);

    hipHostFree(Fx_h);
    hipHostFree(Fy_h);
    hipHostFree(Fz_h);

    hipFree(Fx_d);
    hipFree(Fy_d);
    hipFree(Fz_d);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}

